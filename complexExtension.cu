#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "complexExtension.h"

__host__ __device__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{

	double real = hipCreal(x);
	double imag = hipCimag(x);
	double factor = exp(real);
	return make_hipDoubleComplex(factor * cos(imag), factor * sin(imag));
}

__host__ __device__  hipDoubleComplex cuCcos(hipDoubleComplex x){
	double real = hipCreal(x);
	double imag = hipCimag(x);
	return make_hipDoubleComplex(cos(real)*cosh(imag), -sin(real)*sinh(imag));

}
__host__ __device__ int comparecuDoubleComplex(hipDoubleComplex a, hipDoubleComplex  b){
	if (hipCimag (a) == hipCimag(b) && hipCreal(a) == hipCreal(b) ) return 1;
	return 0;
}

__host__ __device__ hipDoubleComplex cuCsin(hipDoubleComplex x){
	double real = hipCreal(x);
	double imag = hipCimag(x);

	return make_hipDoubleComplex(sin(real)*cosh(imag), cos(real)*sinh(imag));

}
